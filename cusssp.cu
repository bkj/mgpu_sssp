#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wunused-result"

#include "thrust/device_vector.h"

#include <chrono>
#include <queue>
#include <vector>

using namespace std;
using namespace std::chrono;

// --
// Global defs

typedef int Int;
typedef float Real;

// graph
Int n_nodes;
Int n_edges;
Int* indptr;
Int* rindices;
Int* cindices;
Real* data;

// ----------------------------------------------------------------------
// Helpers

__device__ static float atomicMin(float* address, float value) {
  int* addr_as_int = reinterpret_cast<int*>(address);
  int old = *addr_as_int;
  int expected;
  do {
    expected = old;
    old = atomicCAS(addr_as_int, expected, __float_as_int(::fminf(value, __int_as_float(expected))));
  } while (expected != old);
  return __int_as_float(old);
}

template <typename type_t>
void scatter(type_t** out, type_t* h_x, int n, int n_gpus) {
    #pragma omp parallel for num_threads(n_gpus)
    for(int gid = 0; gid < n_gpus; gid++) {
        hipSetDevice(gid);
        
        type_t* d_x;
        hipMalloc(&d_x, n * sizeof(type_t));
        hipMemcpy(d_x, h_x, n * sizeof(type_t),  hipMemcpyHostToDevice);
        
        out[gid] = d_x;
        hipDeviceSynchronize();
    }
    hipSetDevice(0);
}

void load_data(std::string inpath) {
    FILE *ptr;
    ptr = fopen(inpath.c_str(), "rb");

    fread(&n_nodes,   sizeof(Int), 1, ptr);
    fread(&n_nodes,   sizeof(Int), 1, ptr);
    fread(&n_edges,    sizeof(Int), 1, ptr);

    indptr   = (Int*)  malloc(sizeof(Int)  * (n_nodes + 1)  );
    cindices = (Int*)  malloc(sizeof(Int)  * n_edges         );
    rindices = (Int*)  malloc(sizeof(Int)  * n_edges         );
    data     = (Real*) malloc(sizeof(Real) * n_edges         );

    fread(indptr,  sizeof(Int),   n_nodes + 1 , ptr);  // send directy to the memory since thats what the thing is.
    fread(cindices, sizeof(Int),  n_edges      , ptr);
    fread(data,    sizeof(Real),  n_edges      , ptr);
    
    for(Int src = 0; src < n_nodes; src++) {
        for(Int offset = indptr[src]; offset < indptr[src + 1]; offset++) {
            rindices[offset] = src;
        }
    }
}

// ----------------------------------------------------------------------
// CPU implementation

class prioritize {
    public:
        bool operator()(pair<Int, Real> &p1, pair<Int, Real> &p2) {
            return p1.second > p2.second;
        }
};

long long sssp_cpu(Real* dist, Int src) {
    for(Int i = 0; i < n_nodes; i++) dist[i] = std::numeric_limits<Real>::max();
    dist[src] = 0;

    auto t = high_resolution_clock::now();
    priority_queue<pair<Int,Real>, vector<pair<Int,Real>>, prioritize> pq;
    pq.push(make_pair(src, 0));
    
    while(!pq.empty()) {
        pair<Int, Real> curr = pq.top();
        pq.pop();

        Int curr_node  = curr.first;
        Real curr_dist = curr.second;
        if(curr_dist == dist[curr_node]) {
            for(Int offset = indptr[curr_node]; offset < indptr[curr_node + 1]; offset++) {
                Int neib      = cindices[offset];
                Real new_dist = curr_dist + data[offset];
                if(new_dist < dist[neib]) {
                    dist[neib] = new_dist;
                    pq.push(make_pair(neib, new_dist));
                }
            }
        }
    }
    auto elapsed = high_resolution_clock::now() - t;
    return duration_cast<microseconds>(elapsed).count();
}

// ----------------------------------------------------------------------
// GPU implementation

long long sssp_mgpu(Real* h_dist, Int src, Int n_gpus) {    

    // --
    // Setup devices
    
    hipSetDevice(0);
    hipStream_t master_stream;
    hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);

    struct gpu_info {
        hipStream_t stream;
        hipEvent_t  event;
    };
    
    std::vector<gpu_info> infos;
    
    for(int i = 0 ; i < n_gpus ; i++) {
        gpu_info info;
        hipSetDevice(i);
        hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
        hipEventCreate(&info.event);
        infos.push_back(info);
    }
    
    // Enable peer access
    for(int i = 0; i < n_gpus; i++) {
        hipSetDevice(i);
        for(int j = 0; j < n_gpus; j++) {
            if(i == j) continue;
            hipDeviceEnablePeerAccess(j, 0);
        }
    }
    
    hipSetDevice(0);

    // --
    // Setup chunks
    
    Int* starts    = (Int*)malloc(n_gpus * sizeof(Int));
    Int* ends      = (Int*)malloc(n_gpus * sizeof(Int));
    Int chunk_size = (n_edges + n_gpus - 1) / n_gpus;
    for(Int i = 0; i < n_gpus; i++) {
        starts[i] = i * chunk_size;
        ends[i]   = (i + 1) * chunk_size;
    }
    ends[n_gpus - 1] = n_edges;

    // --
    // Setup frontiers
    
    char* h_frontier_in  = (char*)malloc(n_nodes * sizeof(char));
    char* h_frontier_out = (char*)malloc(n_nodes * sizeof(char));
    
    for(Int i = 0; i < n_nodes; i++) h_dist[i]          = std::numeric_limits<Real>::max();
    for(Int i = 0; i < n_nodes; i++) h_frontier_in[i]   = -1;
    for(Int i = 0; i < n_nodes; i++) h_frontier_out[i]  = -1;
    
    h_dist[src]        = 0;
    h_frontier_in[src] = 0;
    
    // Global frontier + dist
    char* g_frontier_in;
    char* g_frontier_out;
    Real* g_dist;
    
    hipMalloc(&g_frontier_in,  n_nodes * sizeof(char));
    hipMalloc(&g_frontier_out, n_nodes * sizeof(char));
    hipMalloc(&g_dist,         n_nodes * sizeof(Real));

    hipMemcpy(g_frontier_in,  h_frontier_in,  n_nodes * sizeof(char),  hipMemcpyHostToDevice);
    hipMemcpy(g_frontier_out, h_frontier_out, n_nodes * sizeof(char),  hipMemcpyHostToDevice);
    hipMemcpy(g_dist,         h_dist,         n_nodes * sizeof(Real),  hipMemcpyHostToDevice);
    
    // Local data, frontier + dist
    Int* all_cindices[n_gpus];
    Int* all_rindices[n_gpus];
    Real* all_data[n_gpus];
    char* all_frontier_in[n_gpus];
    char* all_frontier_out[n_gpus];
    Real* all_dist[n_gpus];

    scatter(all_cindices,     cindices,       n_edges, n_gpus);
    scatter(all_rindices,     rindices,       n_edges, n_gpus);
    scatter(all_data,         data,           n_edges, n_gpus);
    scatter(all_frontier_in,  h_frontier_in,  n_nodes, n_gpus);
    scatter(all_frontier_out, h_frontier_out, n_nodes, n_gpus);
    scatter(all_dist,         h_dist,         n_nodes, n_gpus);

    int iter = 0;
    
    auto t = high_resolution_clock::now();
    while(iter <= 7) { // hardcode number of iterations -- skipping convergence criterionfor now
        
        Int next_iter = iter + 1;
        
        if(iter >= 3 && iter <= 5) {
            
            // Broadcast data to workers
            // Could do this better -- shaped like tree instead of start
            #pragma omp parallel for num_threads(n_gpus)
            for(int gid = 0; gid < n_gpus; gid++) {
                hipSetDevice(gid);
                hipMemcpyAsync(all_frontier_in[gid],  g_frontier_in,  n_nodes * sizeof(char), hipMemcpyDeviceToDevice, infos[gid].stream);
                hipMemcpyAsync(all_dist[gid],         g_dist,         n_nodes * sizeof(Real), hipMemcpyDeviceToDevice, infos[gid].stream);
                hipEventRecord(infos[gid].event, infos[gid].stream);
            }
            for(int gid = 0; gid < n_gpus; gid++)
                hipStreamWaitEvent(master_stream, infos[gid].event, 0);
            hipStreamSynchronize(master_stream);
            
            // Advance
            #pragma omp parallel for num_threads(n_gpus)
            for(int gid = 0; gid < n_gpus; gid++) {
                
                hipSetDevice(gid);
                
                Int* l_cindices      = all_cindices[gid];
                Int* l_rindices      = all_rindices[gid];
                Real* l_data         = all_data[gid];
                char* l_frontier_in  = all_frontier_in[gid];
                char* l_frontier_out = all_frontier_out[gid];
                Real* l_dist         = all_dist[gid];
                
                // Advance
                auto edge_op = [=] __device__(int const& offset) -> void {
                    Int src = l_rindices[offset];
                    Int dst = l_cindices[offset];
                    
                    if(l_frontier_in[src] != iter) return;
                    
                    Real new_dist = l_dist[src] + l_data[offset];     
                    Real old_dist = atomicMin(l_dist + dst, new_dist);
                    if(new_dist < old_dist)
                        l_frontier_out[dst] = next_iter;
                };
                
                thrust::for_each(
                    thrust::cuda::par.on(infos[gid].stream),
                    thrust::make_counting_iterator<Int>(starts[gid]),
                    thrust::make_counting_iterator<Int>(ends[gid]),
                    edge_op
                );
                
                // Merge
                auto merge_op = [=] __device__(int const& dst) -> void {
                    if(l_frontier_out[dst] != next_iter) return;
                    if(g_frontier_out[dst] != next_iter) g_frontier_out[dst] = next_iter;
                    atomicMin(g_dist + dst, l_dist[dst]);
                };
                
                thrust::for_each(
                    thrust::cuda::par.on(infos[gid].stream),
                    thrust::make_counting_iterator<Int>(0),
                    thrust::make_counting_iterator<Int>(n_nodes),
                    merge_op
                );
                
                hipEventRecord(infos[gid].event, infos[gid].stream);
            }
            
            for(int gid = 0; gid < n_gpus; gid++)
                hipStreamWaitEvent(master_stream, infos[gid].event, 0);
            hipStreamSynchronize(master_stream);
        
        } else {
            // Single-GPU mode
            
            hipSetDevice(0);
            
            Int* l_cindices = all_cindices[0];
            Int* l_rindices = all_rindices[0];
            Real* l_data    = all_data[0];

            auto edge_op = [=] __device__(int const& offset) -> void {
                Int src = l_rindices[offset];
                Int dst = l_cindices[offset];
                
                if(g_frontier_in[src] != iter) return; 
                
                Real new_dist = g_dist[src] + l_data[offset];
                Real old_dist = atomicMin(g_dist + dst, new_dist);
                if(new_dist < old_dist)
                    g_frontier_out[dst] = next_iter;
            };
            
            thrust::for_each(
                thrust::cuda::par.on(infos[0].stream),
                thrust::make_counting_iterator<Int>(0),
                thrust::make_counting_iterator<Int>(n_edges),
                edge_op
            );
            
            hipEventRecord(infos[0].event, infos[0].stream);
            hipStreamWaitEvent(master_stream, infos[0].event, 0);
            hipStreamSynchronize(master_stream);
        }
        
        // Swap frontiers
        char* tmp      = g_frontier_in;
        g_frontier_in  = g_frontier_out;
        g_frontier_out = tmp;
                
        iter++;
    }
    
    hipMemcpy(h_dist, g_dist, n_nodes * sizeof(Real), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    auto elapsed = high_resolution_clock::now() - t;
    return duration_cast<microseconds>(elapsed).count();
}


int main(int n_args, char** argument_array) {
    int n_gpus = 1;
    hipGetDeviceCount(&n_gpus);
    
    // ---------------- INPUT ----------------

    load_data(argument_array[1]);

    int src = 0;
    // ---------------- DIJKSTRA ----------------
    
    Real* dijkstra_dist = (Real*)malloc(n_nodes * sizeof(Real));
    auto ms1 = sssp_cpu(dijkstra_dist, src);
    
    // ---------------- FRONTIER ----------------
    
    Real* frontier_dist = (Real*)malloc(n_nodes * sizeof(Real));
    auto ms2 = sssp_mgpu(frontier_dist, src, n_gpus);

    for(Int i = 0; i < 40; i++) std::cout << dijkstra_dist[i] << " ";
    std::cout << std::endl;
    for(Int i = 0; i < 40; i++) std::cout << frontier_dist[i] << " ";
    std::cout << std::endl;

    int n_errors = 0;
    for(Int i = 0; i < n_nodes; i++) {
        if(dijkstra_dist[i] != frontier_dist[i]) n_errors++;
    }
    
    std::cout << "ms1=" << ms1 << " | ms2=" << ms2 << " | n_errors=" << n_errors << std::endl;
    
    return 0;
}
