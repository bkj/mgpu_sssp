#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wunused-result"

#include <cstdlib>  // EXIT_SUCCESS
#include "omp.h"
#include "nvToolsExt.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <thrust/transform_scan.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <chrono>
#include <string.h>
#include <omp.h>

#include <queue>
#include <vector>

using namespace std;
using namespace std::chrono;

// #define VERBOSE
// #define NODE_BALANCED

// --
// Global defs

typedef int Int;
typedef float Real;

// graph
Int n_rows, n_cols, n_nnz;
Int* indptr;
Int* rindices;
Int* indices;
Real* data;

Int n_nodes;
Int n_edges;

__device__ static float atomicMin(float* address, float value) {
  int* addr_as_int = reinterpret_cast<int*>(address);
  int old = *addr_as_int;
  int expected;
  do {
    expected = old;
    old = atomicCAS(addr_as_int, expected, __float_as_int(::fminf(value, __int_as_float(expected))));
  } while (expected != old);
  return __int_as_float(old);
}

// --
// IO

void load_data(std::string inpath) {
    FILE *ptr;
    ptr = fopen(inpath.c_str(), "rb");

    fread(&n_rows,   sizeof(Int), 1, ptr);
    fread(&n_cols,   sizeof(Int), 1, ptr);
    fread(&n_nnz,    sizeof(Int), 1, ptr);

    indptr   = (Int*)  malloc(sizeof(Int)  * (n_rows + 1)  );
    indices  = (Int*)  malloc(sizeof(Int)  * n_nnz         );
    data     = (Real*) malloc(sizeof(Real) * n_nnz         );

    fread(indptr,  sizeof(Int),   n_rows + 1 , ptr);  // send directy to the memory since thats what the thing is.
    fread(indices, sizeof(Int),   n_nnz      , ptr);
    fread(data,    sizeof(Real),  n_nnz      , ptr);

    n_nodes = n_rows;
    n_edges = n_nnz;
    
    rindices = (Int*) malloc(sizeof(Int) * n_nnz);
    for(Int src = 0; src < n_nodes; src++) {
        for(Int offset = indptr[src]; offset < indptr[src + 1]; offset++) {
            rindices[offset] = src;
        }
    }
    
#ifdef VERBOSE
        printf("----------------------------\n");
        printf("n_rows   = %d\n", n_rows);
        printf("n_cols   = %d\n", n_cols);
        printf("n_nnz    = %d\n", n_nnz);
        printf("----------------------------\n");
#endif
}

// --
// Run

class prioritize {
    public:
        bool operator()(pair<Int, Real> &p1, pair<Int, Real> &p2) {
            return p1.second > p2.second;
        }
};

long long dijkstra_sssp(Real* dist, Int src) {
    for(Int i = 0; i < n_nodes; i++) dist[i] = 999.0;
    dist[src] = 0;

    auto t = high_resolution_clock::now();
    priority_queue<pair<Int,Real>, vector<pair<Int,Real>>, prioritize> pq;
    pq.push(make_pair(src, 0));
    
    while(!pq.empty()) {
        pair<Int, Real> curr = pq.top();
        pq.pop();

        Int curr_node  = curr.first;
        Real curr_dist = curr.second;
        if(curr_dist == dist[curr_node]) {
            for(Int offset = indptr[curr_node]; offset < indptr[curr_node + 1]; offset++) {
                Int neib      = indices[offset];
                Real new_dist = curr_dist + data[offset];
                if(new_dist < dist[neib]) {
                    dist[neib] = new_dist;
                    pq.push(make_pair(neib, new_dist));
                }
            }
        }
    }
    auto elapsed = high_resolution_clock::now() - t;
    return duration_cast<microseconds>(elapsed).count();
}

long long frontier_sssp(Real* dist, Int src, Int n_gpus) {
    bool* frontier_in  = (bool*)malloc(n_nodes * sizeof(bool));
    bool* frontier_out = (bool*)malloc(n_nodes * sizeof(bool));
    
    for(Int i = 0; i < n_nodes; i++) dist[i]          = 999.0;
    for(Int i = 0; i < n_nodes; i++) frontier_in[i]   = false;
    for(Int i = 0; i < n_nodes; i++) frontier_out[i]  = false;
    
    dist[src]        = 0;
    frontier_in[src] = true;
    
    int iteration = 0;
    
    // Create chunks
    Int* starts    = (Int*)malloc(n_gpus * sizeof(Int));
    Int* ends      = (Int*)malloc(n_gpus * sizeof(Int));
    Int chunk_size = (n_edges + n_gpus - 1) / n_gpus;
    for(Int i = 0; i < n_gpus; i++) {
        starts[i] = i * chunk_size;
        ends[i]   = (i + 1) * chunk_size;
    }
    ends[n_gpus - 1] = n_edges;

    // Create GPUs
    hipSetDevice(0);
    hipStream_t master_stream;
    hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);

    struct gpu_info {
        hipStream_t stream;
        hipEvent_t  event;
    };
    
    std::vector<gpu_info> infos;
    
    for(int i = 0 ; i < n_gpus ; i++) {
        gpu_info info;
        hipSetDevice(i);
        hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
        hipEventCreate(&info.event);
        infos.push_back(info);
    }
    
    // Enable peer access
    for(int i = 0; i < n_gpus; i++) {
        hipSetDevice(i);
        for(int j = 0; j < n_gpus; j++) {
            if(i == j) continue;
            hipDeviceEnablePeerAccess(j, 0);
        }
    }
    
    hipSetDevice(0);
    
    // Data
    Int* d_indptr;
    Int* d_indices;
    Int* d_rindices;
    Real* d_data;

    hipMallocManaged(&d_indptr,  (n_nodes + 1) * sizeof(Int));
    hipMallocManaged(&d_indices,  n_edges * sizeof(Int));
    hipMallocManaged(&d_rindices, n_edges * sizeof(Int));
    hipMallocManaged(&d_data,     n_edges * sizeof(Real));

    for(int i = 0; i < n_gpus; i++) {
        hipMemAdvise(d_indptr,   (n_nodes + 1) * sizeof(Int), hipMemAdviseSetReadMostly,  i);
        hipMemAdvise(d_indices,  n_edges       * sizeof(Int), hipMemAdviseSetReadMostly,  i);
        hipMemAdvise(d_rindices, n_edges       * sizeof(Int), hipMemAdviseSetReadMostly,  i);
        hipMemAdvise(d_data,     n_edges       * sizeof(Real), hipMemAdviseSetReadMostly, i);
    }
    
    hipMemcpy(d_indptr,   indptr,   (n_nodes + 1) * sizeof(Int), hipMemcpyHostToDevice);
    hipMemcpy(d_indices,  indices,  n_edges * sizeof(Int),       hipMemcpyHostToDevice);
    hipMemcpy(d_rindices, rindices, n_edges * sizeof(Int),       hipMemcpyHostToDevice);
    hipMemcpy(d_data,     data,     n_edges * sizeof(Real),      hipMemcpyHostToDevice);

    // Frontiers
    bool* d_frontier_in;
    bool* d_frontier_out;
    Real* d_dist;
    
    hipMalloc(&d_frontier_in,  n_nodes * sizeof(bool));
    hipMalloc(&d_frontier_out, n_nodes * sizeof(bool));
    hipMalloc(&d_dist,         n_nodes * sizeof(Real));

    hipMemcpy(d_frontier_in,  frontier_in,  n_nodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_frontier_out, frontier_out, n_nodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_dist,         dist,         n_nodes * sizeof(Real), hipMemcpyHostToDevice);

    auto t = high_resolution_clock::now();
    for(int it = 0; it < 5; it++) {
        
#ifdef NODE_BALANCED
        auto node_op = [=] __device__(int const& src) -> bool {
            if(!d_frontier_in[src]) return false;
            
            for(int offset = d_indptr[src]; offset < d_indptr[src + 1]; offset++) {
                Int dst       = d_indices[offset];
                Real new_dist = d_dist[src] + d_data[offset];
                
                if(new_dist < d_dist[dst]) {
                    d_dist[dst]         = new_dist; // false sharing? bad atomics?           
                    d_frontier_out[dst] = true;     // false sharing?
                }
            }
            return false;
        };
        
        thrust::transform(
            thrust::device,
            thrust::make_counting_iterator<int>(0),
            thrust::make_counting_iterator<int>(n_nodes),
            thrust::make_discard_iterator(),
            node_op
        );
#else   

        #pragma omp parallel for num_threads(n_gpus)
        for(int tid = 0; tid < n_gpus; tid++) {
            auto edge_op = [=] __device__(int const& offset) -> bool {
                Int src = d_rindices[offset];
                Int dst = d_indices[offset];
                
                if(!d_frontier_in[src]) return false;
                
                Real new_dist = d_dist[src] + d_data[offset];
                Real old_dist = atomicMin(d_dist + dst, new_dist);
                if(new_dist < old_dist) {
                    d_frontier_out[dst] = true;
                }
                
                return false;
            };

            hipSetDevice(tid);
            thrust::transform(
                thrust::cuda::par.on(infos[tid].stream),
                thrust::make_counting_iterator<int>(starts[tid]),
                thrust::make_counting_iterator<int>(ends[tid]),
                thrust::make_discard_iterator(),
                edge_op
            );
            hipEventRecord(infos[tid].event, infos[tid].stream);
        }
        
        for(int tid = 0; tid < n_gpus; tid++) {
            hipStreamWaitEvent(master_stream, infos[tid].event, 0);
        }
#endif
        
        thrust::fill_n(
            thrust::device,
            d_frontier_in,
            n_nodes,
            false
        );

        bool* tmp      = d_frontier_in;
        d_frontier_in  = d_frontier_out;
        d_frontier_out = tmp;
                
        iteration++;
    }
    
    hipMemcpy(dist, d_dist, n_nodes * sizeof(Real), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    auto elapsed = high_resolution_clock::now() - t;
    return duration_cast<microseconds>(elapsed).count();
}


int main(int n_args, char** argument_array) {
    int n_gpus = 1;
    hipGetDeviceCount(&n_gpus);
    
    // ---------------- INPUT ----------------

    load_data(argument_array[1]);

    int src = 0;
    // ---------------- DIJKSTRA ----------------
    
    Real* dijkstra_dist = (Real*)malloc(n_nodes * sizeof(Real));
    auto ms1 = dijkstra_sssp(dijkstra_dist, src);
    
    // ---------------- FRONTIER ----------------
    
    Real* frontier_dist = (Real*)malloc(n_nodes * sizeof(Real));
    long long ms2;
    for(Int i = 0; i < 5; i++)
        ms2 = frontier_sssp(frontier_dist, src, n_gpus);

    for(Int i = 0; i < 40; i++) std::cout << dijkstra_dist[i] << " ";
    std::cout << std::endl;
    for(Int i = 0; i < 40; i++) std::cout << frontier_dist[i] << " ";
    std::cout << std::endl;

    int n_errors = 0;
    for(Int i = 0; i < n_nodes; i++) {
        if(dijkstra_dist[i] != frontier_dist[i]) n_errors++;
    }
    
    std::cout << "ms1=" << ms1 << " | ms2=" << ms2 << " | n_errors=" << n_errors << std::endl;
    
    return 0;
}
