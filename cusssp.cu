#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wunused-result"

#include <cstdlib>  // EXIT_SUCCESS
#include "omp.h"
#include "nvToolsExt.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <thrust/transform_scan.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <chrono>
#include <string.h>
#include <omp.h>

#include <queue>
#include <vector>

using namespace std;
using namespace std::chrono;

// #define VERBOSE
// #define NODE_BALANCED

// --
// Global defs

typedef int Int;
typedef float Real;

// graph
Int n_rows, n_cols, n_nnz;
Int* indptr;
Int* rindices;
Int* indices;
Real* data;

Int n_nodes;
Int n_edges;

__device__ static float atomicMin(float* address, float value) {
  int* addr_as_int = reinterpret_cast<int*>(address);
  int old = *addr_as_int;
  int expected;
  do {
    expected = old;
    old = atomicCAS(addr_as_int, expected, __float_as_int(::fminf(value, __int_as_float(expected))));
  } while (expected != old);
  return __int_as_float(old);
}

__global__ void edge_kernel(Int start, Int end, Real* d_dist, Int* d_rindices, Int* d_indices, Real* d_data, bool* d_frontier_in, bool* d_frontier_out) {
    Int offset = start + blockIdx.x * blockDim.x + threadIdx.x;
    if(offset >= end) return;
    
    Int src = d_rindices[offset];
    Int dst = d_indices[offset];
    
    if(!d_frontier_in[src]) return;
    
    Real new_dist = d_dist[src] + d_data[offset];
    Real old_dist = atomicMin(d_dist + dst, new_dist);
    if(new_dist < old_dist)
        d_frontier_out[dst] = true;
};

// --
// IO

void load_data(std::string inpath) {
    FILE *ptr;
    ptr = fopen(inpath.c_str(), "rb");

    fread(&n_rows,   sizeof(Int), 1, ptr);
    fread(&n_cols,   sizeof(Int), 1, ptr);
    fread(&n_nnz,    sizeof(Int), 1, ptr);

    indptr   = (Int*)  malloc(sizeof(Int)  * (n_rows + 1)  );
    indices  = (Int*)  malloc(sizeof(Int)  * n_nnz         );
    data     = (Real*) malloc(sizeof(Real) * n_nnz         );

    fread(indptr,  sizeof(Int),   n_rows + 1 , ptr);  // send directy to the memory since thats what the thing is.
    fread(indices, sizeof(Int),   n_nnz      , ptr);
    fread(data,    sizeof(Real),  n_nnz      , ptr);

    n_nodes = n_rows;
    n_edges = n_nnz;
    
    rindices = (Int*) malloc(sizeof(Int) * n_nnz);
    for(Int src = 0; src < n_nodes; src++) {
        for(Int offset = indptr[src]; offset < indptr[src + 1]; offset++) {
            rindices[offset] = src;
        }
    }
    
#ifdef VERBOSE
        printf("----------------------------\n");
        printf("n_rows   = %d\n", n_rows);
        printf("n_cols   = %d\n", n_cols);
        printf("n_nnz    = %d\n", n_nnz);
        printf("----------------------------\n");
#endif
}

// --
// Run

class prioritize {
    public:
        bool operator()(pair<Int, Real> &p1, pair<Int, Real> &p2) {
            return p1.second > p2.second;
        }
};

long long dijkstra_sssp(Real* dist, Int src) {
    for(Int i = 0; i < n_nodes; i++) dist[i] = 999.0;
    dist[src] = 0;

    auto t = high_resolution_clock::now();
    priority_queue<pair<Int,Real>, vector<pair<Int,Real>>, prioritize> pq;
    pq.push(make_pair(src, 0));
    
    while(!pq.empty()) {
        pair<Int, Real> curr = pq.top();
        pq.pop();

        Int curr_node  = curr.first;
        Real curr_dist = curr.second;
        if(curr_dist == dist[curr_node]) {
            for(Int offset = indptr[curr_node]; offset < indptr[curr_node + 1]; offset++) {
                Int neib      = indices[offset];
                Real new_dist = curr_dist + data[offset];
                if(new_dist < dist[neib]) {
                    dist[neib] = new_dist;
                    pq.push(make_pair(neib, new_dist));
                }
            }
        }
    }
    auto elapsed = high_resolution_clock::now() - t;
    return duration_cast<microseconds>(elapsed).count();
}

long long frontier_sssp(Real* dist, Int src, Int n_gpus) {
    bool* frontier_in  = (bool*)malloc(n_nodes * sizeof(bool));
    bool* frontier_out = (bool*)malloc(n_nodes * sizeof(bool));
    
    for(Int i = 0; i < n_nodes; i++) dist[i]          = 999.0;
    for(Int i = 0; i < n_nodes; i++) frontier_in[i]   = false;
    for(Int i = 0; i < n_nodes; i++) frontier_out[i]  = false;
    
    dist[src]        = 0;
    frontier_in[src] = true;
    
    int iteration = 0;
    
    // Create chunks
    Int* starts    = (Int*)malloc(n_gpus * sizeof(Int));
    Int* ends      = (Int*)malloc(n_gpus * sizeof(Int));
    Int chunk_size = (n_edges + n_gpus - 1) / n_gpus;
    for(Int i = 0; i < n_gpus; i++) {
        starts[i] = i * chunk_size;
        ends[i]   = (i + 1) * chunk_size;
    }
    ends[n_gpus - 1] = n_edges;

    // Create GPUs
    hipSetDevice(0);
    hipStream_t master_stream;
    hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);

    struct gpu_info {
        hipStream_t stream;
        hipEvent_t  event;
    };
    
    std::vector<gpu_info> infos;
    
    std::cout << "n_gpus: " << n_gpus << std::endl;
    
    for(int i = 0 ; i < n_gpus ; i++) {
        std::cout << "creating " << i << std::endl;
        gpu_info info;
        hipSetDevice(i);
        hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
        hipEventCreate(&info.event);
        infos.push_back(info);
    }
    
    // Enable peer access
    for(int i = 0; i < n_gpus; i++) {
        hipSetDevice(i);
        for(int j = 0; j < n_gpus; j++) {
            if(i == j) continue;
            hipDeviceEnablePeerAccess(j, 0);
        }
    }
    
    hipSetDevice(0);
    
    // Data
    Int* all_d_indptr[n_gpus];
    Int* all_d_indices[n_gpus];
    Int* all_d_rindices[n_gpus];
    Real* all_d_data[n_gpus];

    for(int i = 0; i < n_gpus; i++) {
        Int* d_indptr;
        Int* d_indices;
        Int* d_rindices;
        Real* d_data;

        hipMalloc(&d_indptr,  (n_nodes + 1) * sizeof(Int));
        hipMalloc(&d_indices,  n_edges * sizeof(Int));
        hipMalloc(&d_rindices, n_edges * sizeof(Int));
        hipMalloc(&d_data,     n_edges * sizeof(Real));
        
        hipMemcpy(d_indptr,   indptr,   (n_nodes + 1) * sizeof(Int), hipMemcpyHostToDevice);
        hipMemcpy(d_indices,  indices,  n_edges * sizeof(Int),       hipMemcpyHostToDevice);
        hipMemcpy(d_rindices, rindices, n_edges * sizeof(Int),       hipMemcpyHostToDevice);
        hipMemcpy(d_data,     data,     n_edges * sizeof(Real),      hipMemcpyHostToDevice);
        
        all_d_indptr[i] = d_indptr;
        all_d_indices[i] = d_indices;
        all_d_rindices[i] = d_rindices;
        all_d_data[i] = d_data;
    }
    
    // for(int i = 0; i < n_gpus; i++) {
    //     hipMemAdvise(d_indptr,   (n_nodes + 1) * sizeof(Int), hipMemAdviseSetReadMostly,  i);
    //     hipMemAdvise(d_indices,  n_edges       * sizeof(Int), hipMemAdviseSetReadMostly,  i);
    //     hipMemAdvise(d_rindices, n_edges       * sizeof(Int), hipMemAdviseSetReadMostly,  i);
    //     hipMemAdvise(d_data,     n_edges       * sizeof(Real), hipMemAdviseSetReadMostly, i);
    // }
    
    // Frontiers
    bool* d_frontier_in;
    bool* d_frontier_out;
    Real* d_dist;
    
    hipMalloc(&d_frontier_in,  n_nodes * sizeof(bool));
    hipMalloc(&d_frontier_out, n_nodes * sizeof(bool));
    hipMalloc(&d_dist,         n_nodes * sizeof(Real));

    hipMemcpy(d_frontier_in,  frontier_in,  n_nodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_frontier_out, frontier_out, n_nodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_dist,         dist,         n_nodes * sizeof(Real), hipMemcpyHostToDevice);

    for(int i = 0 ; i < n_gpus ; i++) hipDeviceSynchronize();

    auto t = high_resolution_clock::now();
    for(int it = 0; it < 1; it++) {

        // auto edge_op = [=] __device__(int const& offset) -> bool {
        //     Int src = d_rindices[offset];
        //     Int dst = d_indices[offset];
            
        //     if(!d_frontier_in[src]) return false;
            
        //     Real new_dist = d_dist[src] + d_data[offset];
        //     Real old_dist = atomicMin(d_dist + dst, new_dist);
        //     if(new_dist < old_dist)
        //         d_frontier_out[dst] = true;
            
        //     return false;
        // };
        
        #pragma omp parallel for num_threads(n_gpus)
        for(int tid = 0; tid < n_gpus; tid++) {
            hipSetDevice(tid);
            // thrust::transform(
            //     thrust::cuda::par.on(infos[tid].stream),
            //     thrust::make_counting_iterator<int>(starts[tid]),
            //     thrust::make_counting_iterator<int>(ends[tid]),
            //     thrust::make_discard_iterator(),
            //     edge_op
            // );
            edge_kernel<<<(n_edges + 255) / 256, 256, 0, infos[tid].stream>>>(
                starts[tid],
                ends[tid],
                d_dist, 
                all_d_rindices[tid], 
                all_d_indices[tid], 
                all_d_data[tid],
                d_frontier_in,
                d_frontier_out
            );
            hipEventRecord(infos[tid].event, infos[tid].stream);
        }
        
        for(int tid = 0; tid < n_gpus; tid++) {
            hipStreamWaitEvent(master_stream, infos[tid].event, 0);
        }
        hipStreamSynchronize(master_stream);
          
        thrust::fill_n(
            thrust::cuda::par.on(infos[0].stream),
            d_frontier_in,
            n_nodes,
            false
        );
        hipEventRecord(infos[0].event, infos[0].stream);
        hipStreamWaitEvent(master_stream, infos[0].event, 0);
        hipStreamSynchronize(master_stream);

        bool* tmp      = d_frontier_in;
        d_frontier_in  = d_frontier_out;
        d_frontier_out = tmp;
                
        iteration++;
    }
    
    hipMemcpy(dist, d_dist, n_nodes * sizeof(Real), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    auto elapsed = high_resolution_clock::now() - t;
    return duration_cast<microseconds>(elapsed).count();
}


int main(int n_args, char** argument_array) {
    int n_gpus = 1;
    hipGetDeviceCount(&n_gpus);
    
    // ---------------- INPUT ----------------

    load_data(argument_array[1]);

    int src = 0;
    // ---------------- DIJKSTRA ----------------
    
    Real* dijkstra_dist = (Real*)malloc(n_nodes * sizeof(Real));
    auto ms1 = dijkstra_sssp(dijkstra_dist, src);
    
    // ---------------- FRONTIER ----------------
    
    Real* frontier_dist = (Real*)malloc(n_nodes * sizeof(Real));
    auto ms2 = frontier_sssp(frontier_dist, src, n_gpus);

    for(Int i = 0; i < 40; i++) std::cout << dijkstra_dist[i] << " ";
    std::cout << std::endl;
    for(Int i = 0; i < 40; i++) std::cout << frontier_dist[i] << " ";
    std::cout << std::endl;

    int n_errors = 0;
    for(Int i = 0; i < n_nodes; i++) {
        if(dijkstra_dist[i] != frontier_dist[i]) n_errors++;
    }
    
    std::cout << "ms1=" << ms1 << " | ms2=" << ms2 << " | n_errors=" << n_errors << std::endl;
    
    return 0;
}
