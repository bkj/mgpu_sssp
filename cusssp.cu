#include "hip/hip_runtime.h"
#pragma GCC diagnostic ignored "-Wunused-result"

#include <cstdlib>  // EXIT_SUCCESS
#include "omp.h"
#include "nvToolsExt.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

#include <thrust/transform_scan.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>

#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <chrono>
#include <string.h>
#include <omp.h>
#include <queue>
#include <vector>

#include "timer.hxx"

using namespace std;
using namespace std::chrono;

// --
// Global defs

typedef int Int;
typedef float Real;

// graph
Int n_rows, n_cols, n_nnz;
Int* indptr;
Int* rindices;
Int* indices;
Real* data;

Int n_nodes;
Int n_edges;

__device__ static float atomicMin(float* address, float value) {
  int* addr_as_int = reinterpret_cast<int*>(address);
  int old = *addr_as_int;
  int expected;
  do {
    expected = old;
    old = atomicCAS(addr_as_int, expected,
                      __float_as_int(::fminf(value, __int_as_float(expected))));
  } while (expected != old);
  return __int_as_float(old);
}

// --
// IO

void load_data(std::string inpath) {
    FILE *ptr;
    ptr = fopen(inpath.c_str(), "rb");

    fread(&n_rows,   sizeof(Int), 1, ptr);
    fread(&n_cols,   sizeof(Int), 1, ptr);
    fread(&n_nnz,    sizeof(Int), 1, ptr);

    indptr   = (Int*)  malloc(sizeof(Int)  * (n_rows + 1)  );
    indices  = (Int*)  malloc(sizeof(Int)  * n_nnz         );
    data     = (Real*) malloc(sizeof(Real) * n_nnz         );

    fread(indptr,  sizeof(Int),   n_rows + 1 , ptr);  // send directy to the memory since thats what the thing is.
    fread(indices, sizeof(Int),   n_nnz      , ptr);
    fread(data,    sizeof(Real),  n_nnz      , ptr);

    n_nodes = n_rows;
    n_edges = n_nnz;
    
    rindices = (Int*) malloc(sizeof(Int) * n_nnz);
    for(Int src = 0; src < n_nodes; src++) {
        for(Int offset = indptr[src]; offset < indptr[src + 1]; offset++) {
            rindices[offset] = src;
        }
    }
}

// --
// Run

class prioritize {
    public:
        bool operator()(pair<Int, Real> &p1, pair<Int, Real> &p2) {
            return p1.second > p2.second;
        }
};

void dijkstra_sssp(Real* dist, Int src) {
    for(Int i = 0; i < n_nodes; i++) dist[i] = 999.0;
    dist[src] = 0;

    priority_queue<pair<Int,Real>, vector<pair<Int,Real>>, prioritize> pq;
    pq.push(make_pair(src, 0));

    while(!pq.empty()) {
        pair<Int, Real> curr = pq.top();
        pq.pop();

        Int curr_node  = curr.first;
        Real curr_dist = curr.second;
        if(curr_dist == dist[curr_node]) {
            for(Int offset = indptr[curr_node]; offset < indptr[curr_node + 1]; offset++) {
                Int neib      = indices[offset];
                Real new_dist = curr_dist + data[offset];
                if(new_dist < dist[neib]) {
                    dist[neib] = new_dist;
                    pq.push(make_pair(neib, new_dist));
                }
            }
        }
    }
}

void advance(Real* dist, bool* frontier_in, bool* frontier_out, Int start, Int end) {
    for(Int src = start; src < end; src++) {
        if(!frontier_in[src]) continue;
        frontier_in[src] = false;
        
        for(int offset = indptr[src]; offset < indptr[src + 1]; offset++) {
            Int dst       = indices[offset];
            Real new_dist = dist[src] + data[offset];
            
            if(new_dist < dist[dst]) {
                dist[dst]         = new_dist; // false sharing? bad atomics?           
                frontier_out[dst] = true;     // false sharing?
            }
        }
    }
}

long long frontier_sssp(Real* dist, Int src, Int n_threads) {
    
    // --
    // Data from host to device
    
    Int* d_indptr;
    Int* d_indices;
    Int* d_rindices;
    Real* d_data;

    hipMalloc(&d_indptr,  (n_nodes + 1) * sizeof(Int));
    hipMalloc(&d_indices,  n_edges * sizeof(Int));
    hipMalloc(&d_rindices, n_edges * sizeof(Int));
    hipMalloc(&d_data,     n_edges * sizeof(Real));

    hipMemcpy(d_indptr,   indptr,   (n_nodes + 1) * sizeof(Int), hipMemcpyHostToDevice);
    hipMemcpy(d_indices,  indices,  n_edges * sizeof(Int),       hipMemcpyHostToDevice);
    hipMemcpy(d_rindices, rindices, n_edges * sizeof(Int),       hipMemcpyHostToDevice);
    hipMemcpy(d_data,     data,     n_edges * sizeof(Real),      hipMemcpyHostToDevice);
    
    // --
    // Setup problem
    
    bool* frontier_in  = (bool*)malloc(n_nodes * sizeof(bool));
    bool* frontier_out = (bool*)malloc(n_nodes * sizeof(bool));
    
    for(Int i = 0; i < n_nodes; i++) dist[i]          = 999.0;
    for(Int i = 0; i < n_nodes; i++) frontier_in[i]   = false;
    for(Int i = 0; i < n_nodes; i++) frontier_out[i]  = false;
    
    dist[src]        = 0;
    frontier_in[src] = true;
    
    int iteration = 0;
    
    // Frontiers
    bool* d_frontier_in;
    bool* d_frontier_out;
    Real* d_dist;
    
    hipMalloc(&d_frontier_in,  n_nodes * sizeof(bool));
    hipMalloc(&d_frontier_out, n_nodes * sizeof(bool));
    hipMalloc(&d_dist,         n_nodes * sizeof(Real));

    hipMemcpy(d_frontier_in,  frontier_in,  n_nodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_frontier_out, frontier_out, n_nodes * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_dist,         dist,         n_nodes * sizeof(Real), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    auto t = high_resolution_clock::now();
    
    cuda_timer_t my_timer;
    my_timer.start();
    
    while(true) {

        auto edge_op = [=] __device__(int const& offset) -> bool {
            Int src = d_rindices[offset];
            Int dst = d_indices[offset];
            
            if(!d_frontier_in[src]) return false;
            
            Real new_dist = d_dist[src] + d_data[offset];
            Real old_dist = atomicMin(d_dist + dst, new_dist);
            if(new_dist < old_dist)
                d_frontier_out[dst] = true;
            
            return false;
        };

        thrust::transform(
            thrust::device,
            thrust::make_counting_iterator<int>(0),
            thrust::make_counting_iterator<int>(n_edges),
            thrust::make_discard_iterator(),
            edge_op
        );

        bool* tmp      = d_frontier_in;
        d_frontier_in  = d_frontier_out;
        d_frontier_out = tmp;
        
        thrust::fill_n(
            thrust::device,
            d_frontier_out,
            n_nodes,
            false
        );
        
        // Convergence criterion
        auto keep_going = thrust::reduce(
            thrust::device,
            d_frontier_in + 0,
            d_frontier_in + n_nodes
        );
        if(keep_going == 0) break; 
        
        iteration++;
    }
    
    hipMemcpy(dist, d_dist, n_nodes * sizeof(Real), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    float ms = my_timer.end();
    std::cout << "cuda ms: " << ms << std::endl; 
    
    auto elapsed = high_resolution_clock::now() - t;
    return duration_cast<microseconds>(elapsed).count();
}


int main(int n_args, char** argument_array) {
    
    // ---------------- INPUT ----------------

    load_data(argument_array[1]);

    int src = 0;
    // ---------------- DIJKSTRA ----------------
    
    Real* dijkstra_dist = (Real*)malloc(n_nodes * sizeof(Real));
    auto t1       = high_resolution_clock::now();
    dijkstra_sssp(dijkstra_dist, src);
    auto elapsed1 = high_resolution_clock::now() - t1;
    long long ms1 = duration_cast<microseconds>(elapsed1).count();
    
    // ---------------- FRONTIER ----------------
    
    Real* frontier_dist = (Real*)malloc(n_nodes * sizeof(Real));
    auto ms2 = frontier_sssp(frontier_dist, src, 1);

    for(Int i = 0; i < 40; i++) std::cout << dijkstra_dist[i] << " ";
    std::cout << std::endl;
    for(Int i = 0; i < 40; i++) std::cout << frontier_dist[i] << " ";
    std::cout << std::endl;

    int n_errors = 0;
    for(Int i = 0; i < n_nodes; i++) {
        if(dijkstra_dist[i] != frontier_dist[i]) n_errors++;
    }
    
    std::cout << "ms1=" << ms1 << " | ms2=" << ms2 << " | n_errors=" << n_errors << std::endl;
    
    return 0;
}
